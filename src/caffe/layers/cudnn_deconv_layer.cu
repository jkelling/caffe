
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>
#include "caffe/layers/cudnn_deconv_layer.hpp"

namespace caffe {

__global__ void sync_deconv_groups() {}

template <typename Dtype>
void CuDNNDeconvolutionLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  VLOG(1) << "  " << this->layer_param_.name() << " Forward GPU" << std::endl;
  const Dtype* weight = this->blobs_[0]->gpu_data();
  for (int i = 0; i < bottom.size(); ++i) {
    for (int n = 0; n < this->num_; ++n) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* top_data = top[i]->mutable_gpu_data();
      // Forward through cuDNN in parallel over groups.
      for (int g = 0; g < this->group_; g++) {
        // Filters.
        CUDNN_CHECK(cudnnConvolutionBackwardData(
              handle_[g],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight + this->weight_offset_ * g,
              bottom_descs_[i], bottom_data + bottom_offset_ * g + n * this->bottom_dim_,
              conv_descs_[i],
              bwd_data_algo_[i], workspace[g],
              workspace_bwd_data_sizes_[i],
              cudnn::dataType<Dtype>::zero,
              top_descs_[i], top_data + top_offset_ * g + n * this->top_dim_));
        // Bias.
        if (this->bias_term_) {
            const Dtype* bias_data = this->blobs_[1]->gpu_data();
            CUDNN_CHECK(cudnnAddTensor(handle_[g],
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_data + bias_offset_ * g,
              cudnn::dataType<Dtype>::one,
              top_descs_[i], top_data + top_offset_ * g + n * this->top_dim_));
        }
      }
      // Synchronize the work across groups, each of which went into its own
      // stream, by launching an empty kernel into the default (null) stream.
      // NOLINT_NEXT_LINE(whitespace/operators)
      sync_deconv_groups<<<1, 1>>>();
    }
  }
}

template <typename Dtype>
void CuDNNDeconvolutionLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  VLOG(1) << "  " << this->layer_param_.name() << " Backward GPU"
          << std::endl;
  const Dtype* weight = NULL;
  Dtype* weight_diff = NULL;
  for (int i = 0; i < top.size(); ++i) {
    if (this->param_propagate_down_[0]) {
      weight = this->blobs_[0]->gpu_data();
      weight_diff = this->blobs_[0]->mutable_gpu_diff();
    }
    Dtype* bias_diff = NULL;
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      bias_diff = this->blobs_[1]->mutable_gpu_diff();
    }
    const Dtype* top_diff = top[i]->gpu_diff();
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
    // Backward through cuDNN in parallel over groups and gradients.
    for (int g = 0; g < this->group_; g++) {
      // Gradient w.r.t. bias.
      if (this->bias_term_ && this->param_propagate_down_[1]) {
      for (int n = 0; n < this->num_; ++n) {
          CUDNN_CHECK(cudnnConvolutionBackwardBias(handle_[0*this->group_ + g],
            cudnn::dataType<Dtype>::one,
            top_descs_[i],  top_diff + top_offset_ * g + n * this->top_dim_,
            cudnn::dataType<Dtype>::one,
            bias_desc_, bias_diff + bias_offset_ * g));
          }
      }

      // Gradient w.r.t. weights. Note that we will accumulate diffs.
      if (this->param_propagate_down_[0] || propagate_down[i]) {
        for (int n = 0; n < this->num_; ++n) {
          if (this->param_propagate_down_[0]) {
            CUDNN_CHECK(cudnnConvolutionBackwardFilter(
              handle_[1*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              top_descs_[i],    top_diff + top_offset_ * g + n * this->top_dim_,
              bottom_descs_[i], bottom_data + bottom_offset_ * g +
              n * this->bottom_dim_, conv_descs_[i],
              bwd_filter_algo_[i], workspace[1*this->group_ + g],
              workspace_bwd_filter_sizes_[i],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight_diff + this->weight_offset_ * g));
          }
          // Gradient w.r.t. bottom data.
          if (propagate_down[i]) {
            if (weight == NULL) {
              weight = this->blobs_[0]->gpu_data();
            }
            CUDNN_CHECK(cudnnConvolutionForward(handle_[2*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              top_descs_[i], top_diff + top_offset_ * g + n * this->top_dim_,
              filter_desc_, weight + this->weight_offset_ * g,
              conv_descs_[i],
              fwd_algo_[i], workspace[2*this->group_ + g],
              workspace_fwd_sizes_[i],
              cudnn::dataType<Dtype>::zero,
              bottom_descs_[i], bottom_diff + bottom_offset_ * g +
              n * this->bottom_dim_));
          }
        }
      }
    }
    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_deconv_groups<<<1, 1>>>();
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNDeconvolutionLayer);

}  // namespace caffe
#endif
